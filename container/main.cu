#include "hip/hip_runtime.h"
#include <iostream>
#include <chrono>
#include <sched.h>

#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>

using namespace std::chrono;

void random_ints(int *a, int N) {
    int i;
    for (i = 0; i < N; ++i) a[i] = rand();
}

__global__ void kernel(int N, int *a, int *b, int *c)
{
    const int index = blockIdx.x * blockDim.x + threadIdx.x; 
    if(index < N)
        c[index] = a[index] + b[index];
}

int main(int argc, char **argv) {

    const int N = argc > 1 ? atoi(argv[1]) : 10;

    // host arrays
    int *a = new int[N];
    int *b = new int[N];
    int *c = new int[N];

    // initialize inputs
    random_ints(a, N);
    random_ints(b, N);

    int *d_a, *d_b, *d_c;
    hipMalloc(&d_a, N * sizeof(int));
    hipMalloc(&d_b, N * sizeof(int));
    hipMalloc(&d_c, N * sizeof(int));

    hipMemcpy(d_a, a, N * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_b, b, N * sizeof(int), hipMemcpyHostToDevice);

    // do work
    const int threads = 32;
    const int blocks = N/threads + 1;
    kernel<<<blocks,threads>>> (N, d_a, d_b, d_c);

    for (int i = 0; i < N; i++) 
        printf("c[%d] = %d\n",i,c[i]);

    // cleanup
    delete[] a;
    delete[] b;
    delete[] c;

    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_c);

    return 0;
}
